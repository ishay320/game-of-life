#include <hip/hip_runtime.h>
#include <signal.h>
#include <stdio.h>
#include <unistd.h>

#define BOARD_WIDTH 30
#define BOARD_HEIGHT 20

bool should_run = true;
void signal_handler(int) { should_run = false; }

typedef enum _States
{
    DEAD,
    ALIVE,
    STATES_LEN
} States;

#define POS(arr, i, j, width) arr[(i)*width + (j)]
#define cudaCheckErrors(msg)                                                                                        \
    do                                                                                                              \
    {                                                                                                               \
        hipError_t __err = hipGetLastError();                                                                     \
        if (__err != hipSuccess)                                                                                   \
        {                                                                                                           \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg, hipGetErrorString(__err), __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n");                                                             \
            exit(1);                                                                                                \
        }                                                                                                           \
    } while (0)

void print_board(States *board, size_t width, size_t height)
{
    for (size_t i = 0; i < height; i++)
    {
        for (size_t j = 0; j < width; j++)
        {
            switch (POS(board, i, j, width))
            {
                case 0:
                    putchar(' ');
                    break;
                case 1:
                    putchar('#');
                    break;

                default:
                    printf("\n%s:%d %s ERROR: unreachable code, state: %d\n", __FILE__, __LINE__, __FUNCTION__, board[i * width + j]);
                    exit(1);
                    break;
            }
            putchar(' ');
        }
        putchar('\n');
    }
}

#define ROUND_POS(arr, i, j, width, height) POS(arr, ((i + height) % height), ((j + width) % width), width)
__device__ uint8_t get_neighbors(States *board, size_t width, size_t height, size_t i, size_t j)
{
    uint8_t count = 0;

    for (int row = i - 1; row <= (int)i + 1; row++)
    {
        for (int column = j - 1; column <= (int)j + 1; column++)
        {
            if (!(row == (int)i && column == (int)j) && ROUND_POS(board, row, column, width, height) == 1)
            {
                count++;
            }
        }
    }
    return count;
}

__global__ void step(States *board, States *board_out, size_t width, size_t height)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= (width * height))
    {
        printf("%d, idx %d", width * height, idx);
        return;
    }

    int j = idx % width;
    int i = (idx - j) / width;

    uint8_t neighbors = get_neighbors(board, width, height, i, j);

    if (POS(board, i, j, width) == ALIVE && (2 == neighbors || neighbors == 3))
    {
        POS(board_out, i, j, width) = ALIVE;
    }
    else if (neighbors == 3)
    {
        POS(board_out, i, j, width) = ALIVE;
    }
    else
    {
        POS(board_out, i, j, width) = DEAD;
    }
}

void clear_screen()
{
    const char *CLEAR_SCREEN_ANSI = "\e[1;1H\e[2J";
    write(STDOUT_FILENO, CLEAR_SCREEN_ANSI, 11);
}

void switch_board(States **board_a, States **board_b)
{
    States *tmp = *board_b;
    *board_b    = *board_a;
    *board_a    = tmp;
}

int main(void)
{
    // Signal for ending the loop peacefully
    if (signal(SIGINT, signal_handler) == SIG_ERR)
    {
        perror("signal");
        return 1;
    }

    size_t size       = BOARD_WIDTH * BOARD_HEIGHT;
    States *board_cpu = (States *)malloc(size * sizeof(States));

    States *board_gpu;
    hipMalloc((void **)&board_gpu, size * sizeof(States));
    cudaCheckErrors("hipMalloc fail");
    States *board_gpu_out;
    hipMalloc((void **)&board_gpu_out, size * sizeof(States));
    cudaCheckErrors("hipMalloc fail");

    // Initialize host array
    for (size_t i = 0; i < BOARD_WIDTH * BOARD_HEIGHT; i++)
    {
        board_cpu[i] = DEAD;
    }

    {  // R-Pentomino
        POS(board_cpu, (BOARD_HEIGHT / 2) + 0, (BOARD_WIDTH / 2) + 1, BOARD_WIDTH) = ALIVE;
        POS(board_cpu, (BOARD_HEIGHT / 2) + 0, (BOARD_WIDTH / 2) + 0, BOARD_WIDTH) = ALIVE;
        POS(board_cpu, (BOARD_HEIGHT / 2) + 1, (BOARD_WIDTH / 2) + 0, BOARD_WIDTH) = ALIVE;
        POS(board_cpu, (BOARD_HEIGHT / 2) + 2, (BOARD_WIDTH / 2) + 0, BOARD_WIDTH) = ALIVE;
        POS(board_cpu, (BOARD_HEIGHT / 2) + 1, (BOARD_WIDTH / 2) - 1, BOARD_WIDTH) = ALIVE;
    }

    // Copy to CUDA device
    hipMemcpy(board_gpu, board_cpu, size * sizeof(States), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy 1 fail");

    // Size of blocks and threads
    int block_size = 1;
    int n_blocks   = size / block_size + (size % block_size == 0 ? 0 : 1);
    step<<<n_blocks, block_size>>>(board_gpu, board_gpu_out, BOARD_WIDTH, BOARD_HEIGHT);
    hipDeviceSynchronize();
    cudaCheckErrors("kernel fail");

    while (should_run)
    {
        // Retrieve result from device and store it in host array
        hipDeviceSynchronize();
        hipMemcpy(board_cpu, board_gpu_out, size * sizeof(States), hipMemcpyDeviceToHost);
        cudaCheckErrors("hipMemcpy 2 fail");

        step<<<n_blocks, block_size>>>(board_gpu, board_gpu_out, BOARD_WIDTH, BOARD_HEIGHT);
        cudaCheckErrors("kernel fail");

        // Swap boards
        States *tmp   = board_gpu_out;
        board_gpu_out = board_gpu;
        board_gpu     = tmp;

        // Print results
        clear_screen();
        print_board(board_cpu, BOARD_WIDTH, BOARD_HEIGHT);

        usleep(100000);
    }

    // Cleanup
    free(board_cpu);
    hipFree(board_gpu);
    hipFree(board_gpu_out);
}